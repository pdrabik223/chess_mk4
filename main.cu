
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel()
{
    printf("Hello from block %u, thread %u\n", blockIdx.x, threadIdx.x);
}

int main()
{
    kernel<<<8, 16>>>();
    hipError_t error = hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
